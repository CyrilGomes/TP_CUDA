/*
//@HEADER
// ************************************************************************
//
//                        Kokkos v. 2.0
//              Copyright (2014) Sandia Corporation
//
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions Contact  H. Carter Edwards (hcedwar@sandia.gov)
//
// ************************************************************************
//@HEADER
*/


#include <hip/hip_runtime.h>
#include <limits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>
#include <assert.h> 
#include <cmath>
#include <omp.h>
#include <iostream>
#include <fstream>
#include <iomanip>

using namespace std;
void checkSizes( int &N, int &M, int &S, int &nrepeat );
void write_perf_csv(int n, int m, int repeat, double runtime);

static long tpb = 1;
static long num_blocks = 10;

__device__ void multiplyVectors(double* a, double* b, int numline, int nbcol, float *sum) {
  int index = numline*nbcol;

  int threadi = threadIdx.x;
  int stride = blockDim.x;
  for (int j=threadi;j< nbcol; j+= stride){
    atomicAdd(sum, a[index+j] * b[j]);

  }


}
__global__ void mulKer(double* A, double* x,double* y, int nblines, int nbcol,  float* result,float* sum){


    int i = blockIdx.x;
    printf("sum before : %f\n",*sum);

    multiplyVectors(A,x, i, nbcol, sum) ;
    printf("sum after : %f\n",*sum);

    __syncthreads();
    if(threadIdx.x == 0)
      atomicAdd(result, (*sum)* y[i]);


}


int main( int argc, char* argv[] )
{
  int N = 4096;         // number of rows 2^12
  int M = 1024;         // number of columns 2^10
  int S = 4096*1024;         // total size 2^22
  int nrepeat = 100;  // number of repeats of the test

  // Read command line arguments.
  for ( int i = 0; i < argc; i++ ) {
    if ( ( strcmp( argv[ i ], "-N" ) == 0 ) || ( strcmp( argv[ i ], "-Rows" ) == 0 ) ) {
      N = pow( 2, atoi( argv[ ++i ] ) );
      printf( "  User N is %d\n", N );
    }
    else if ( ( strcmp( argv[ i ], "-M" ) == 0 ) || ( strcmp( argv[ i ], "-Columns" ) == 0 ) ) {
      M = pow( 2, atof( argv[ ++i ] ) );
      printf( "  User M is %d\n", M );
    }
    else if ( ( strcmp( argv[ i ], "-S" ) == 0 ) || ( strcmp( argv[ i ], "-Size" ) == 0 ) ) {
      S = pow( 2, atof( argv[ ++i ] ) );
      printf( "  User S is %d\n", S );
    }
    else if ( ( strcmp( argv[ i ], "-tpb" ) == 0 )) {
      tpb = atoi(argv[ ++i ]);
      printf( "  User TPB is %d\n",  tpb);
    }
    else if ( ( strcmp( argv[ i ], "-B" ) == 0 )) {
      num_blocks = atoi( argv[ ++i ] );
      printf( "  User Num Blocks is %d\n", num_blocks );
    }
    else if ( strcmp( argv[ i ], "-nrepeat" ) == 0 ) {
      nrepeat = atoi( argv[ ++i ] );
    }
    else if ( ( strcmp( argv[ i ], "-h" ) == 0 ) || ( strcmp( argv[ i ], "-help" ) == 0 ) ) {
      printf( "  y^T*A*x Options:\n" );
      printf( "  -Rows (-N) <int>:      exponent num, determines number of rows 2^num (default: 2^12 = 4096)\n" );
      printf( "  -Columns (-M) <int>:   exponent num, determines number of columns 2^num (default: 2^10 = 1024)\n" );
      printf( "  -Size (-S) <int>:      exponent num, determines total matrix size 2^num (default: 2^22 = 4096*1024 )\n" );
      printf( "  -nrepeat <int>:        number of repetitions (default: 100)\n" );
      printf( "  -help (-h):            print this message\n\n" );
      exit( 1 );
    }
  }
  S = N*M;
  // Check sizes.
  checkSizes( N, M, S, nrepeat );

  // Allocate x,y,A
  double* y = new double[N];
  double* x = new double[M];
  double* A = new double[S];
  float result_h = 0;
  float sum=0;

  // Allocate x,y,A for device
  double* y_d;
  double* x_d;
  double* A_d;
  float* result_d;
  float* sum_d;

  // Initialize y vector to 1.
  for (int i = 0; i<N; i++){
    y[i] = 1;
  }

  // Initialize x vector to 1.
  for (int i = 0; i<M; i++){
    x[i] = 1;
  }

  // Initialize A matrix, you can use a 1D index if you want a flat structure (i.e. a 1D array) e.g. j*M+i is the same than [j][i]
  for (int i = 0; i<S; i++){
      A[i] = 1;
  }

  // Timer products.
  struct timeval begin, end;

  gettimeofday( &begin, NULL );

  hipMalloc((void **) &sum_d, sizeof(float) );
  hipMalloc((void **) &result_d, sizeof(float) );
  hipMalloc((void **) &x_d, sizeof(double)*M);
  hipMalloc((void **) &y_d, sizeof(double)*N);
  hipMalloc((void **) &A_d, sizeof(double)*S);

  hipMemcpy(A_d, A, sizeof(double)*S, hipMemcpyHostToDevice);
  hipMemcpy(x_d, x, sizeof(double)*M, hipMemcpyHostToDevice);
  hipMemcpy(y_d, y, sizeof(double)*N, hipMemcpyHostToDevice);




	  
  for ( int repeat = 0; repeat < nrepeat; repeat++ ) {
    // For each line i
        // Multiply the i lines with the vector x 
        // Sum the results of the previous step into a single variable
        // Multiply the result of the previous step with the i value of vector y
        // Sum the results of the previous step into a single variable (result)

    result_h = 0;
    sum = 0;
    hipMemcpy(sum_d, &sum, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(result_d, &result_h, sizeof(float), hipMemcpyHostToDevice);

    num_blocks = 2;
    mulKer<<<num_blocks, tpb>>>(A_d, x_d, y_d , N, M, result_d,sum_d);

    hipMemcpy(&result_h, result_d, sizeof(float), hipMemcpyDeviceToHost);


    // Output result.
    if ( repeat == ( nrepeat - 1 ) ) {
      printf( "  Computed result for %d x %d is %lf\n", N, M, result_h );
    }

    const double solution = (double) N * (double) M;

    if ( result_h != solution ) {
      printf( "  Error: result( %lf ) != solution( %lf )\n", result_h, solution );
    }

    
  }
  hipFree(result_d);
  hipFree(sum_d);
  gettimeofday( &end, NULL );

  // Calculate time.
  //double time = timer.seconds();
  double time = 1.0 * ( end.tv_sec - begin.tv_sec ) +
                1.0e-6 * ( end.tv_usec - begin.tv_usec );

  // Calculate bandwidth.
  // Each matrix A row (each of length M) is read once.
  // The x vector (of length M) is read N times.
  // The y vector (of length N) is read once.
  // double Gbytes = 1.0e-9 * double( sizeof(double) * ( 2 * M * N + N ) );
  double Gbytes = 1.0e-9 * double( sizeof(double) * ( M + M * N + N ) );

  // Print results (problem size, time and bandwidth in GB/s).
  printf( "  N( %d ) M( %d ) nrepeat ( %d ) problem( %g MB ) time( %g s ) bandwidth( %g GB/s )\n",
          N, M, nrepeat, Gbytes * 1000, time, Gbytes * nrepeat / time );

  //write_perf_csv(N, M, nrepeat, time); 
  std::free(A);
  std::free(y);
  std::free(x);

  return 0;
}

void checkSizes( int &N, int &M, int &S, int &nrepeat ) {
  // If S is undefined and N or M is undefined, set S to 2^22 or the bigger of N and M.
  if ( S == -1 && ( N == -1 || M == -1 ) ) {
    S = pow( 2, 22 );
    if ( S < N ) S = N;
    if ( S < M ) S = M;
  }

  // If S is undefined and both N and M are defined, set S = N * M.
  if ( S == -1 ) S = N * M;

  // If both N and M are undefined, fix row length to the smaller of S and 2^10 = 1024.
  if ( N == -1 && M == -1 ) {
    if ( S > 1024 ) {
      M = 1024;
    }
    else {
      M = S;
    }
  }

  // If only M is undefined, set it.
  if ( M == -1 ) M = S / N;

  // If N is undefined, set it.
  if ( N == -1 ) N = S / M;

  printf( "  Total size S = %d N = %d M = %d\n", S, N, M );

  // Check sizes.
  if ( ( S < 0 ) || ( N < 0 ) || ( M < 0 ) || ( nrepeat < 0 ) ) {
    printf( "  Sizes must be greater than 0.\n" );
    exit( 1 );
  }

  if ( ( N * M ) != S ) {
    printf( "  N * M != S\n" );
    exit( 1 );
  }
}


void write_perf_csv(int n, int m, int repeat, double runtime){
  ofstream myfile;
  myfile.open ("stats_part2.csv", ios_base::app);
  myfile.precision(8);
  myfile <<n<<"," << m << ","<< repeat << "," << runtime << "\n";

  myfile.close();
}
