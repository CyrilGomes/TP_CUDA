/*

This program will numerically compute the integral of

                  4/(1+x*x) 
				  
from 0 to 1.  The value of this integral is pi -- which 
is great since it gives us an easy way to check the answer.

History: Written by Tim Mattson, 11/1999.
         Modified/extended by Jonathan Rouzaud-Cornabas, 10/2022
*/


#include <hip/hip_runtime.h>
#include <limits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>

static long num_steps = 100000000;
static long thread_per_block = 1;
static long steps_per_thread = 64;
double step;

__global__ void compute_pi(float* pi, long num_steps){
      int i;
      double x;
      __shared__ float sum;

      if(threadIdx.x == 0){
        sum = 0;
      }
      int threadi = threadIdx.x + blockIdx.x * blockDim.x;
      int stride = blockDim.x * gridDim.x;
      double step = 1.0/(double) num_steps;

      for (i=threadi;i< num_steps; i+= stride){
        x = (i-0.5)*step;
        atomicAdd(&sum, 4.0/(1.0+x*x));
      }
      __syncthreads();

      if(threadIdx.x == 0){
        atomicAdd(pi, sum);
      }
}

int main (int argc, char** argv)
{
    
      // Read command line arguments.
      for ( int i = 0; i < argc; i++ ) {
        if ( ( strcmp( argv[ i ], "-N" ) == 0 ) || ( strcmp( argv[ i ], "-num_steps" ) == 0 ) ) {
            num_steps = atol( argv[ ++i ] );
            printf( "  User num_steps is %ld\n", num_steps );
        }
        if ( ( strcmp( argv[ i ], "-tpb" ) == 0 )) {
            thread_per_block = atol( argv[ ++i ] );
            printf( "  User thread per block is %ld\n", thread_per_block );
        } 
        if ( ( strcmp( argv[ i ], "-spt" ) == 0 )) {
            steps_per_thread = atol( argv[ ++i ] );
            printf( "  User steps per thread is %ld\n", steps_per_thread );
        } 
        else if ( ( strcmp( argv[ i ], "-h" ) == 0 ) || ( strcmp( argv[ i ], "-help" ) == 0 ) ) {
            printf( "  Pi Options:\n" );
            printf( "  -num_steps (-N) <int>:      Number of steps to compute Pi (by default 100000000)\n" );
            printf( "  -help (-h):            print this message\n\n" );
            exit( 1 );
        }
      }
      
    float pi_h = 0;

    float* pi_d;
    hipMalloc((void **) &pi_d, sizeof(float) );
    hipMemcpy(pi_d, &pi_h, sizeof(float), hipMemcpyHostToDevice);

	  
    step = 1.0/(double) num_steps;

    int num_blocks = num_steps/(thread_per_block*steps_per_thread) + 1;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    compute_pi<<<num_blocks, thread_per_block>>>(pi_d, num_steps);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;

    hipMemcpy(&pi_h, pi_d, sizeof(float), hipMemcpyDeviceToHost);
    hipEventElapsedTime(&elapsedTime, start, stop);

	  pi_h = step * pi_h;

    
    printf("\n pi with %ld steps is %lf in %lf s\n",num_steps,pi_h,elapsedTime/1000.0);

    hipFree(pi_d);
}