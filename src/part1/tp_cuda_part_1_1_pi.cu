/*

This program will numerically compute the integral of

                  4/(1+x*x) 
				  
from 0 to 1.  The value of this integral is pi -- which 
is great since it gives us an easy way to check the answer.

History: Written by Tim Mattson, 11/1999.
         Modified/extended by Jonathan Rouzaud-Cornabas, 10/2022
*/


#include <hip/hip_runtime.h>
#include <limits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>
#include <iostream>
#include <fstream>
#include <iomanip>

using namespace std;

static long num_steps = 100000000;
static long thread_per_block = 1;
static long steps_per_thread = 64;
double step;

void write_perf_csv(int thread_per_block, int step_per_thread,int num_steps, double runtime){
  ofstream myfile;
  myfile.open ("stats_part1.csv", ios_base::app);
  myfile.precision(8);
  myfile <<"1_1 basic"<<"," << thread_per_block << ","<< step_per_thread <<","<< num_steps << "," << runtime << "\n";

  myfile.close();
}

__global__ void compute_pi(float* pi, long num_steps){
      int i;
      double x, sum = 0.0;
      int threadi = threadIdx.x + blockIdx.x * blockDim.x;
      int stride = blockDim.x * gridDim.x;
      double step = 1.0/(double) num_steps;

      for (i=threadi;i< num_steps; i+= stride){
        x = (i-0.5)*step;
        sum = sum + 4.0/(1.0+x*x);
      }

      atomicAdd(pi, sum);


}

int main (int argc, char** argv)
{
    
      // Read command line arguments.
      for ( int i = 0; i < argc; i++ ) {
        if ( ( strcmp( argv[ i ], "-N" ) == 0 ) || ( strcmp( argv[ i ], "-num_steps" ) == 0 ) ) {
            num_steps = atol( argv[ ++i ] );
            printf( "  User num_steps is %ld\n", num_steps );
        }
        if ( ( strcmp( argv[ i ], "-tpb" ) == 0 )) {
            thread_per_block = atol( argv[ ++i ] );
            printf( "  User thread per block is %ld\n", thread_per_block );
        } 
        if ( ( strcmp( argv[ i ], "-spt" ) == 0 )) {
            steps_per_thread = atol( argv[ ++i ] );
            printf( "  User steps per thread is %ld\n", steps_per_thread );
        } 
        else if ( ( strcmp( argv[ i ], "-h" ) == 0 ) || ( strcmp( argv[ i ], "-help" ) == 0 ) ) {
            printf( "  Pi Options:\n" );
            printf( "  -num_steps (-N) <int>:      Number of steps to compute Pi (by default 100000000)\n" );
            printf( "  -help (-h):            print this message\n\n" );
            exit( 1 );
        }
      }
      
    float pi_h = 0;

    float* pi_d;
    hipMalloc((void **) &pi_d, sizeof(float) );
    hipMemcpy(pi_d, &pi_h, sizeof(float), hipMemcpyHostToDevice);

	  
    step = 1.0/(double) num_steps;

    int num_blocks = num_steps/(thread_per_block*steps_per_thread) + 1;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    printf("%d", num_blocks);
    compute_pi<<<num_blocks, thread_per_block>>>(pi_d, num_steps);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;

    hipMemcpy(&pi_h, pi_d, sizeof(float), hipMemcpyDeviceToHost);
    hipEventElapsedTime(&elapsedTime, start, stop);

	  pi_h = step * pi_h;

    
    printf("\n pi with %ld steps is %lf in %lf s\n",num_steps,pi_h,elapsedTime/1000.0);
    write_perf_csv(thread_per_block,steps_per_thread,num_steps, elapsedTime/1000.0);
    hipFree(pi_d);
}
